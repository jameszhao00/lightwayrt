#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#include "util.h"
#include "bitmap_image.hpp"
#include "assert.h"

#define CATCH_CONFIG_RUNNER
#include "catch.hpp"

#include "validate_importance_sampling.h"

struct Vec3Buffer
{
	float* x;
	float* y;
	float* z;
	void init(int size) 
	{
		CUDA_CHECK_RETURN(hipMalloc(&x, sizeof(float) * size));
		CUDA_CHECK_RETURN(hipMalloc(&y, sizeof(float) * size));
		CUDA_CHECK_RETURN(hipMalloc(&z, sizeof(float) * size));
	}
	GPU_CPU v3 get(int idx) const
	{
		return v3(x[idx], y[idx], z[idx]);
	}
	GPU_CPU void set(int idx, const v3& v)
	{
		x[idx] = v.x; y[idx] = v.y; z[idx] = v.z;
	}
};
struct Pass
{
	Pass(int iteration_idx, int num_iterations, int num_bounces) 
		: iteration_idx(iteration_idx), num_iterations(num_iterations), num_bounces(num_bounces) { }
	int iteration_idx;
	int num_iterations;
	int num_bounces;
};
surface<void, cudaSurfaceType2D> output_surf;
GPU_ENTRY void gfx_kernel(Vec3Buffer buffer, const Camera* camera, const Scene* scene, const Pass* pass, int width, int height
#ifdef LW_CPU
	, ref::glm::uvec2 xy
#endif
	) {
	ref::glm::uvec2 screen_size(width, height);
		
#ifndef LW_CPU
	ref::glm::uvec2 xy = screen_xy();
#endif
	if(ref::glm::any(ref::glm::greaterThan(xy, screen_size - ref::glm::uvec2(1)))) return;
	int linid = xy.y * width + xy.x;
	
	color value(0,0,0);
	ray<World> ray0 = camera_ray(*camera, xy, screen_size);
	for(int iteration_idx = pass->iteration_idx; iteration_idx < (pass->iteration_idx + pass->num_iterations); iteration_idx++)
	{
		bool use_implicit_light = true;
		ray<World> eye_ray = ray0;
		color throughput(1,1,1);
		for(int bounce_idx = 0; bounce_idx < pass->num_bounces; bounce_idx++)
		{
			Hit<World> hit;
			if(eye_ray.intersect(*scene, &hit, use_implicit_light))
			{
				if(hit.material.emission.is_black())
				{
					if(!hit.material.is_specular)
					{
						RandomPair u = rand2(RandomKey(xy), RandomCounter(iteration_idx, pass->num_bounces + bounce_idx));
						color inv_light_pdf;
						position<World> light_pos = scene->sample_light(hit.position, u, &inv_light_pdf);
						direction<World> light_dir(hit.position, light_pos);

						if(!ray<World>(hit.position, light_dir)
							.offset_by(RAY_EPSILON)
							.intersect_shadow(*scene, light_pos))
						{
							value = value + throughput 
								* clamp01(dot(light_dir, hit.normal)) 
								* inv_light_pdf 
								* hit.material.brdf();
						}

						use_implicit_light = false;
					}
					else
					{
						use_implicit_light = true;
					}
				}
				else
				{
					value = value + throughput * hit.material.emission;
					break;
				}			

				if(bounce_idx != pass->num_bounces - 1)
				{
					if(hit.material.is_specular)
					{
						throughput = throughput * hit.material.albedo;
						eye_ray = ray<World>(hit.position, eye_ray.dir.reflect(hit.normal))
							.offset_by(RAY_EPSILON);
					}
					else
					{
						RandomPair u = rand2(RandomKey(xy), RandomCounter(iteration_idx, bounce_idx));
						InverseProjectedPdf ip_pdf;
						direction<World> wi = sampleUniformHemi(hit.normal, u, &ip_pdf);
						throughput = throughput * ip_pdf * hit.material.brdf();
						eye_ray = ray<World>(hit.position, wi).offset_by(RAY_EPSILON);
					}
				}
			}
			else
			{
				break;
			}

		}
	}
	color existing = buffer.get(linid);
	float existing_weight = (float)pass->iteration_idx / (pass->iteration_idx + pass->num_iterations);
	color combined = (value / (float)pass->num_iterations) 
		* (1 - existing_weight) + existing * (existing_weight);
	buffer.set(linid, combined);
	color combined_tonemapped = combined / (combined + color(1,1,1));
	surf2Dwrite(make_float4(combined_tonemapped.x, combined_tonemapped.y, combined_tonemapped.z, 1), output_surf, xy.x*sizeof(float4), xy.y);
}

void Kernel::setup(hipGraphicsResource* output, int width, int height)
{
	framebuffer_resource = output;
	CUDA_CHECK_RETURN(hipMalloc((void**) &camera_ptr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &scene_ptr, sizeof(Scene)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &pass_ptr, sizeof(Pass)));
	//CUDA_CHECK_RETURN(hipMalloc((void**) &buffer, sizeof(ref::glm::vec4) * width * height));
	buffer = new Vec3Buffer();
	buffer->init(width * height);
}
void Kernel::execute(int iteration_idx, int iterations, int bounces, int width, int height)
{	
	Pass pass(iteration_idx, iterations, bounces);
	Camera camera(position<World>(0,7,-5), position<World>(0,0,1));
	Scene scene;

	CUDA_CHECK_RETURN(hipMemcpy(camera_ptr, &camera, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(scene_ptr, &scene, sizeof(Scene), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(pass_ptr, &pass, sizeof(Pass), hipMemcpyHostToDevice));

	hipArray_t framebuffer_ptr;
	
	CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &framebuffer_resource));
	CUDA_CHECK_RETURN(hipGraphicsSubResourceGetMappedArray(&framebuffer_ptr, framebuffer_resource, 0, 0));
	CUDA_CHECK_RETURN(cudaBindSurfaceToArray(output_surf, framebuffer_ptr));
	dim3 threadPerBlock(16, 16, 1);
	dim3 blocks((unsigned int)ceil(width / (float)threadPerBlock.x), 
		(unsigned int)ceil(height / (float)threadPerBlock.y), 1);	
	gfx_kernel<<<blocks, threadPerBlock>>>(*buffer, (Camera*)camera_ptr, (Scene*)scene_ptr, (Pass*)pass_ptr, width, height);
	CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &framebuffer_resource));
}
/*
const int WIDTH = 600;
const int HEIGHT = 600;

int main(int argc, char* const argv[]) {
#ifdef LW_UNIT_TEST
	Catch::Main( argc, argv );
#else
	ref::glm::vec4 *d = NULL;
	Camera *camera_ptr = NULL;
	Scene *scene_ptr= NULL;
	ref::glm::vec4* odata = new ref::glm::vec4[WIDTH * HEIGHT];
	Pass* pass_ptr = nullptr;
	Pass pass(0, 600, 4);
	Camera camera(position<World>(0,7,-5), position<World>(0,0,1));
	Scene scene;
	CUDA_CHECK_RETURN(hipMalloc((void**) &d, sizeof(ref::glm::vec4) * WIDTH * HEIGHT));
	CUDA_CHECK_RETURN(hipMalloc((void**) &camera_ptr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &scene_ptr, sizeof(Scene)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &pass_ptr, sizeof(Pass)));
	CUDA_CHECK_RETURN(hipMemcpy(camera_ptr, &camera, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(scene_ptr, &scene, sizeof(Scene), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(pass_ptr, &pass, sizeof(Pass), hipMemcpyHostToDevice));
	dim3 threadPerBlock(8, 8, 1);
	dim3 blocks((unsigned int)ceil(WIDTH / (float)threadPerBlock.x), (unsigned int)ceil(HEIGHT / (float)threadPerBlock.y), 1);
	gfx_kernel<<<blocks, threadPerBlock>>>(d, camera_ptr, scene_ptr, pass_ptr, WIDTH, HEIGHT);

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipMemcpy((void*)odata, (void*)d, sizeof(float4) * WIDTH * HEIGHT, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree((void*) d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	bitmap_image image(WIDTH, HEIGHT);
	for(int j = 0; j < HEIGHT; j++)
	{
		for(int i = 0; i < WIDTH; i++)
		{
			ref::glm::vec4 col = odata[j * WIDTH + i];
			col /= 1.f + col;
			col *= 255;
			ref::glm::uvec4 ucolor(col);
			image.set_pixel(i, j, ucolor.x, ucolor.y, ucolor.z);
		}
	}
	image.save_image("out.bmp");
	
#endif
	return 0;
}
*/
#ifdef LW_UNIT_TEST
//tests
TEST_CASE("camera/camera_ray", "standard camera_ray") 
{
	{
		Camera camera(position<World>(0.f,0.f,0.f), position<World>(0, 0, -100));
		ref::glm::uvec2 screen_size(5, 5);
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(2, 2), screen_size);
			REQUIRE(close_to(ray.origin.z, -1));
		}
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(0, 0), screen_size);
			REQUIRE(ray.origin.x < 0);
			REQUIRE(ray.origin.y > 0);	
			REQUIRE(close_to(ray.origin.z, -1));
		}
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(4, 4), screen_size);
			REQUIRE(ray.origin.x > 0);
			REQUIRE(ray.origin.y < 0);	
			REQUIRE(close_to(ray.origin.z, -1));
		}
	}
}

#include <random>
TEST_CASE("diffuse/sample_uniform", "sample hemi") 
{	
	std::mt19937 rng;
	std::uniform_real_distribution<float> normalized_dist(0, 1);
	
	auto b =[&]() -> RandomPair { 
		return RandomPair(normalized_dist(rng), normalized_dist(rng));	
	};
	{
		auto a = [&](RandomPair u, InverseProjectedPdf* inv_pdf) -> direction<World> {
			return direction<World>(sampleUniformHemi(direction<World>(0, 0, 1), ref::glm::vec2(u), (float*)inv_pdf));			
		};
		auto invPdfFunc = [](NormalizedSphericalCS cs)-> float {
			return 2 * PI * cos(cs.x);
		};
		validate_importance_sampling<10, 10>(a, b, invPdfFunc, 1000000);
	}
	{
		auto a = [&](RandomPair u, InverseProjectedPdf* inv_pdf) -> direction<World> {
			return direction<World>(sampleCosWeightedHemi(direction<World>(0, 0, 1), ref::glm::vec2(u), (float*)inv_pdf));			
		};
		auto invPdfFunc = [](NormalizedSphericalCS cs)-> float {
			return PI;// / cos(cs.x);
		};
		validate_importance_sampling<10, 10>(a, b, invPdfFunc, 1000000);
	}
}


TEST_CASE("intersect/intersect_plane", "hits plane everywhere") 
{
	InfiniteHorizontalPlane plane(0, Material(color(1,1,1), color(), false));
	{
		ray<World> ray(position<World>(0,1,0), direction<World>(0,-1,0));
		Hit<World> hit;
		REQUIRE(ray.intersect_plane(plane, &hit));
		REQUIRE(hit.t == 1);
		REQUIRE(hit.normal.y == 1.f);
	}
}

#endif