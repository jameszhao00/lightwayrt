#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "kernel.h"
#include "util.h"
#include "bitmap_image.hpp"
#include "assert.h"

#define CATCH_CONFIG_RUNNER
#include "catch.hpp"

#include "validate_importance_sampling.h"

struct Vec3Buffer
{
	float* x;
	float* y;
	float* z;
	void init(int size) 
	{
		CUDA_CHECK_RETURN(hipMalloc(&x, sizeof(float) * size));
		CUDA_CHECK_RETURN(hipMalloc(&y, sizeof(float) * size));
		CUDA_CHECK_RETURN(hipMalloc(&z, sizeof(float) * size));
	}
	GPU void elementwise_atomic_add(int idx, const v3& v3)
	{
		atomicAdd(x + idx, v3.x);
		atomicAdd(y + idx, v3.y);
		atomicAdd(z + idx, v3.z);
	}
	GPU void elementwise_atomic_add(const ref::glm::uvec2& xy, int width, const v3& v3)
	{
		return elementwise_atomic_add(xy.y * width + xy.x, v3);
	}
	GPU float area(ref::glm::vec2 a, ref::glm::vec2 b)
	{
		return abs(a.x - b.x) * abs(a.y - b.y);
	}
	GPU void elementwise_atomic_add(ref::glm::vec2 xy, int width, const v3& v3)
	{
		auto top_left = ref::glm::vec2(floor(xy.x), floor(xy.y));
		/*
		auto bot_left = top_left + ref::glm::vec2(0, 1);
		auto bot_right = top_left + ref::glm::vec2(1, 1);
		auto top_right = top_left + ref::glm::vec2(1, 0);

		float top_left_area = area(top_left, xy);;
		float bot_left_area = area(bot_left, xy);;
		float bot_right_area = area(bot_right, xy);;
		float top_right_area = 1 - top_left_area - bot_left_area - bot_right_area;
		*/
		elementwise_atomic_add(ref::glm::uvec2(top_left), width, v3);
		//elementwise_atomic_add(ref::glm::uvec2(top_left), width, v3_mul(v3, bot_right_area));
		//elementwise_atomic_add(ref::glm::uvec2(bot_left), width, v3_mul(v3, top_right_area));
		//elementwise_atomic_add(ref::glm::uvec2(bot_right), width, v3_mul(v3, top_left_area));
		//elementwise_atomic_add(ref::glm::uvec2(top_right), width, v3_mul(v3, bot_left_area));
	}
	GPU_CPU v3 get(int idx) const
	{
		return v3(x[idx], y[idx], z[idx]);
	}
	GPU_CPU void set(int idx, const v3& v)
	{
		x[idx] = v.x; y[idx] = v.y; z[idx] = v.z;
	}
};
struct Pass
{
	Pass(int iteration_idx, int num_iterations, int num_bounces, bool bdpt_debug) 
		: iteration_idx(iteration_idx), num_iterations(num_iterations), num_bounces(num_bounces), bdpt_debug(bdpt_debug) { }
	int iteration_idx;
	int num_iterations;
	int num_bounces;
	bool bdpt_debug;
};
GPU_CPU ref::glm::vec2 world_to_screen(position<World> world, 
	const Camera& camera, 
	int width, int height,
	bool* in_bounds,
	ref::glm::vec2& ndc)
{
	auto pos_view = camera.view * ref::glm::vec4(to_glm(world), 1);
	auto pos_clip = camera.proj * pos_view;
	auto ndc_4 = pos_clip / pos_clip.w;
	*in_bounds = ndc_4.x > -1 && ndc_4.y > -1 && ndc_4.x < 1 && ndc_4.y < 1 && ndc_4.z > -1 && ndc_4.z < 1;
	ndc = ref::glm::vec2(ndc_4);
	return ref::glm::vec2(ref::glm::floor(
		(ref::glm::vec2(ndc) * ref::glm::vec2(.5, -.5) + ref::glm::vec2(.5, .5)) * ref::glm::vec2(width, height)));
}
struct Random
{
	GPU_CPU __forceinline__ Random(RandomKey key, RandomCounter base_counter) : key(key), counter(base_counter) { }
	__device__ __forceinline__ RandomPair next2() 
	{
		counter.y++;
		return rand2(key, counter);
	}
	RandomKey key;
	RandomCounter counter;
};

#ifndef LW_UNIT_TEST
surface<void, cudaSurfaceType2D> output_surf;
GPU_ENTRY void transfer_image(Vec3Buffer new_buffer, Vec3Buffer existing_buffer, const Pass* pass, int width, int height)
{
	ref::glm::uvec2 screen_size(width, height);		
#ifndef LW_CPU
	ref::glm::uvec2 xy = screen_xy();
#endif
	if(ref::glm::any(ref::glm::greaterThan(xy, screen_size - ref::glm::uvec2(1)))) return;
	int linid = xy.y * width + xy.x;

	color existing = existing_buffer.get(linid);
	float existing_weight = (float)pass->iteration_idx / (pass->iteration_idx + pass->num_iterations);
	color combined = (color(new_buffer.get(linid)) / (float)pass->num_iterations) * (1 - existing_weight);
	if(pass->iteration_idx > 0)
	{
		combined = combined + existing * (existing_weight);
	}
	new_buffer.set(linid, v3(0,0,0));
	
	existing_buffer.set(linid, combined);
	color combined_tonemapped = combined  / (combined + color(1,1,1));
	surf2Dwrite(make_float4(combined_tonemapped.x, combined_tonemapped.y, combined_tonemapped.z, 1), output_surf, xy.x*sizeof(float4), xy.y);
	
}
GPU_CPU ref::glm::vec2 component_image_position(int width, int height, int eye_verts_count, int light_verts_count, int component_size,
	int original_x, int original_y)
{
	int center_x = width / 2;
	int total_verts = eye_verts_count + light_verts_count; //starts at 3
	int total_components = total_verts + 1; //4 images at 3 verts
	int y_idx = total_verts - 3; //implicit path with length=1 = 2 verts...
	int x_idx = light_verts_count;
	int y = component_size * y_idx;
	int x = center_x - (float)total_components / 2.f * component_size + eye_verts_count * component_size;
	if(light_verts_count == 0) x += 20;
	return ref::glm::vec2(x, y) 
		+ ref::glm::vec2(ref::glm::vec2((float)original_x / width, (float)original_y / height) * ref::glm::vec2(component_size, component_size));
}
GPU_CPU color connection_throughput(const Hit<World>& light, const Hit<World>& eye, const Scene& scene)
{
	if(light.material.type == eSpecular || eye.material.type == eSpecular) return color(0,0,0);

	ray<World> shadow(light.position, eye.position);
	if(shadow.offset_by(RAY_EPSILON).intersect_shadow(scene, eye.position)) return color(0,0,0);
		
	offset<World> disp = eye.position - light.position;
	float d = disp.length();
	direction<World> dir(light.position, eye.position);
	float cos_light = clamp01(dot(dir, light.normal));
	float cos_eye = clamp01(-dot(dir, eye.normal));
	float g = cos_light * cos_eye / (d * d);
	return g;
}
GPU void store_bdpt_debug(Vec3Buffer& buffer, const color& value, const int width, const int height, 
	int ev_count, int lv_count, ref::glm::vec2 xy)
{
	int component_size = 250;
	ref::glm::vec2 component_xy = component_image_position(width, height, ev_count, lv_count, component_size, xy.x, xy.y);
	color add = value * (float)(component_size * component_size) / (width * height);
	buffer.elementwise_atomic_add(component_xy, width, add);
}
GPU void extend_bdpt(Random& rng, const Hit<World>& hit, ray<World>* path_ray, color* throughput)
{
	direction<World> wi;
	if(hit.material.type != eSpecular)
	{
		InverseProjectedPdf ippdf;
		wi = sampleCosWeightedHemi(hit.normal, rng.next2(), &ippdf);					
		*throughput = *throughput * ippdf;
	}
	else
	{
		wi = path_ray->dir.reflect(hit.normal);
		*throughput = *throughput * hit.material.specular.albedo;
	}
	*path_ray = ray<World>(hit.position, wi)
					.offset_by(RAY_EPSILON);
}
GPU_ENTRY void gfx_kernel(Vec3Buffer buffer, const Camera* camera, const Scene* scene, const Pass* pass, int width, int height
#ifdef LW_CPU
//	, ref::glm::uvec2 xy
#endif
	) {
	ref::glm::uvec2 screen_size(width, height);
		
#ifndef LW_CPU
	ref::glm::uvec2 xy = screen_xy();
#endif
	if(ref::glm::any(ref::glm::greaterThan(xy, screen_size - ref::glm::uvec2(1)))) return;
	int linid = xy.y * width + xy.x;
	color summed(0,0,0);
	{
		float a = powf(2 * tanf(0.5 * (camera->fovy / 180) * PI), 2);
		for(int iteration_idx = pass->iteration_idx; iteration_idx < (pass->iteration_idx + pass->num_iterations); 
			iteration_idx++)
		{
			Random rng(xy, RandomCounter(iteration_idx, 0));
			Hit<World> light_vertex;
			color light_throughput;
			float light_spatial_ipdf;
			light_vertex.position = sample_sphere_light(scene->sphere_lights[0], rng.next2(), &light_throughput);
			light_vertex.normal = direction<World>(scene->sphere_lights[0].origin, light_vertex.position);
			light_vertex.material = scene->sphere_lights[0].material;
		
			ray<World> light_ray;
			for(int light_vertex_idx = 0; light_vertex_idx < pass->num_bounces + 1; light_vertex_idx++)
			{			
				//direct connect with eye
				if(light_vertex.material.type != eSpecular)
				{
					bool in_bounds;
					ref::glm::vec2 ndc;
					ref::glm::vec2 uv = world_to_screen(light_vertex.position, *camera, width, height, &in_bounds, ndc);
					if(in_bounds)
					{
						ray<World> light_to_eye_shadow_ray = ray<World>(light_vertex.position, camera->eye)
							.offset_by(RAY_EPSILON);
						if(!light_to_eye_shadow_ray.intersect_shadow(*scene, camera->eye))
						{						
							float costheta_shadow_ev = clamp01(-dot(light_to_eye_shadow_ray.dir, camera->forward));
							float costheta_shadow_lv = clamp01(dot(light_to_eye_shadow_ray.dir, light_vertex.normal));
							float d = (light_vertex.position - camera->eye).length();
							float g = costheta_shadow_ev * costheta_shadow_lv / (d * d);
							float we = 1
								/ (a * costheta_shadow_ev * costheta_shadow_ev * costheta_shadow_ev);
							int variations = 2;
							color addition = light_throughput * light_vertex.material.brdf() 
								* g * we / (float)variations;
							if(pass->bdpt_debug)
							{
								store_bdpt_debug(buffer, addition, width, height, 1, light_vertex_idx + 1, uv);
							}
							else
							{								
								buffer.elementwise_atomic_add(uv, width, addition);
							}
						}
					}
				}
				if(light_vertex_idx < pass->num_bounces)
				{
					/* HACK... should divide by PI for vertex 0 */
					light_throughput = light_throughput * light_vertex.material.brdf();
					extend_bdpt(rng, light_vertex, &light_ray, &light_throughput);
					if(!light_ray.intersect(*scene, &light_vertex)) break;
				}
			}
		}
	}
	//eye
	{		
		color summed(0,0,0);
		
		for(int iteration_idx = pass->iteration_idx; iteration_idx < (pass->iteration_idx + pass->num_iterations);
			iteration_idx++)
		{			
			Random rng(xy, RandomCounter(iteration_idx, 1000));
			color value(0,0,0);
			color eye_throughput(1,1,1);
			ray<World> ray0 = camera_ray(*camera, xy, screen_size);
			Hit<World> eye_vertex_1;
			if(!ray0.intersect(*scene, &eye_vertex_1)) return;
			Hit<World> eye_vertex = eye_vertex_1;
			ray<World> eye_ray = ray0;
			for(int eye_vertex_idx = 1; eye_vertex_idx < pass->num_bounces + 1; eye_vertex_idx++)
			{				
				int variations = 2;				
				color addition(0,0,0);
				if(eye_vertex.material.type == eEmissive) //last vertex is implied to be specular
				{
					addition = eye_vertex.material.emissive.emission * eye_throughput / (float)(variations);
				}
				else if(eye_vertex.material.type == eDiffuse)
				{
					color light_spatial_throughput;
					Hit<World> light = 
						scene->sample_light(eye_vertex.position, rng.next2(), &light_spatial_throughput);
												
					color light_throughput = light_spatial_throughput
						* connection_throughput(light, eye_vertex, *scene);
					
					addition = eye_vertex.material.brdf() * light_throughput * eye_throughput / (float)(variations);
					
				}
				if(!addition.is_black())
				{
					if(pass->bdpt_debug)
					{
						store_bdpt_debug(buffer, addition, width, height, eye_vertex_idx + 1, 
							eye_vertex.material.type == eEmissive ? 0 : 1, ref::glm::vec2(xy));
					}
					else
					{	
						value = value + addition;
					}
				}
				if(eye_vertex.material.type == eEmissive) break;
				if(eye_vertex_idx < pass->num_bounces)
				{
					eye_throughput = eye_throughput * eye_vertex.material.brdf();
					extend_bdpt(rng, eye_vertex, &eye_ray, &eye_throughput);

					if(!eye_ray.intersect(*scene, &eye_vertex, eye_vertex.material.type == eSpecular)) break;
				}
			}
			summed = summed + value;
		}
		
		if(!pass->bdpt_debug)
		{
			buffer.set(linid, summed);
		}
	}
}
void Kernel::setup(hipGraphicsResource* output, int width, int height)
{
	framebuffer_resource = output;
	CUDA_CHECK_RETURN(hipMalloc((void**) &camera_ptr, sizeof(Camera)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &scene_ptr, sizeof(Scene)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &pass_ptr, sizeof(Pass)));
	//CUDA_CHECK_RETURN(hipMalloc((void**) &buffer, sizeof(ref::glm::vec4) * width * height));
	new_buffer = new Vec3Buffer();
	new_buffer->init(width * height);
	existing_buffer = new Vec3Buffer();
	existing_buffer->init(width * height);
}
void Kernel::execute(int iteration_idx, int iterations, int bounces, int width, int height, bool bdpt_debug)
{	
	Pass pass(iteration_idx, iterations, bounces, bdpt_debug);
	Camera camera(position<World>(0,9,-7), position<World>(0,0,1), bdpt_debug ? 1 : (float)width/height);
	Scene scene;

	CUDA_CHECK_RETURN(hipMemcpy(camera_ptr, &camera, sizeof(Camera), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(scene_ptr, &scene, sizeof(Scene), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(pass_ptr, &pass, sizeof(Pass), hipMemcpyHostToDevice));

	dim3 threadPerBlock(16, 16, 1);
	dim3 blocks((unsigned int)ceil(width / (float)threadPerBlock.x), 
		(unsigned int)ceil(height / (float)threadPerBlock.y), 1);	
	gfx_kernel<<<blocks, threadPerBlock>>>(*new_buffer, (Camera*)camera_ptr, 
		(Scene*)scene_ptr, (Pass*)pass_ptr, width, height);


	
	hipArray_t framebuffer_ptr;
	
	CUDA_CHECK_RETURN(hipGraphicsMapResources(1, &framebuffer_resource));
	CUDA_CHECK_RETURN(hipGraphicsSubResourceGetMappedArray(&framebuffer_ptr, framebuffer_resource, 0, 0));
	CUDA_CHECK_RETURN(cudaBindSurfaceToArray(output_surf, framebuffer_ptr));

	transfer_image<<<blocks, threadPerBlock>>>(*new_buffer, 
		*existing_buffer, (Pass*)pass_ptr, width, height);
	CUDA_CHECK_RETURN(hipGraphicsUnmapResources(1, &framebuffer_resource));
}
#endif
#ifdef LW_UNIT_TEST
//tests
TEST_CASE("camera/camera_ray", "standard camera_ray") 
{
	{
		Camera camera(position<World>(0.f,0.f,0.f), position<World>(0, 0, -100));
		ref::glm::uvec2 screen_size(5, 5);
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(2, 2), screen_size);
			REQUIRE(close_to(ray.origin.z, -1));
		}
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(0, 0), screen_size);
			REQUIRE(ray.origin.x < 0);
			REQUIRE(ray.origin.y > 0);	
			REQUIRE(close_to(ray.origin.z, -1));
		}
		{
			ray<World> ray = camera_ray(camera, ref::glm::uvec2(4, 4), screen_size);
			REQUIRE(ray.origin.x > 0);
			REQUIRE(ray.origin.y < 0);	
			REQUIRE(close_to(ray.origin.z, -1));
		}
	}
}

#include <random>
TEST_CASE("diffuse/sample_uniform", "sample hemi") 
{	
	std::mt19937 rng;
	std::uniform_real_distribution<float> normalized_dist(0, 1);
	
	auto b =[&]() -> RandomPair { 
		return RandomPair(normalized_dist(rng), normalized_dist(rng));	
	};
	{
		auto a = [&](RandomPair u, InverseProjectedPdf* inv_pdf) -> direction<World> {
			return direction<World>(sampleUniformHemi(direction<World>(0, 0, 1), ref::glm::vec2(u), (float*)inv_pdf));			
		};
		auto invPdfFunc = [](NormalizedSphericalCS cs)-> float {
			return 2 * PI * cos(cs.x);
		};
		validate_importance_sampling<10, 10>(a, b, invPdfFunc, 1000000);
	}
	{
		auto a = [&](RandomPair u, InverseProjectedPdf* inv_pdf) -> direction<World> {
			return direction<World>(sampleCosWeightedHemi(direction<World>(0, 0, 1), ref::glm::vec2(u), (float*)inv_pdf));			
		};
		auto invPdfFunc = [](NormalizedSphericalCS cs)-> float {
			return PI;// / cos(cs.x);
		};
		validate_importance_sampling<10, 10>(a, b, invPdfFunc, 1000000);
	}
}


TEST_CASE("intersect/intersect_plane", "hits plane everywhere") 
{
	InfiniteHorizontalPlane plane(0, Material(color(1,1,1), color(), false));
	{
		ray<World> ray(position<World>(0,1,0), direction<World>(0,-1,0));
		Hit<World> hit;
		REQUIRE(ray.intersect_plane(plane, &hit));
		REQUIRE(hit.t == 1);
		REQUIRE(hit.normal.y == 1.f);
	}
}
TEST_CASE("math/color", "color math works") 
{
	REQUIRE(all_equal(color(1,1,1) / 2, color(0.5f, 0.5f, 0.5f)));
	//REQUIRE(all_equal(color(1,1,1) + 2, color(3,3,3)));
	REQUIRE(all_equal(color(1,1,1) / color(2,2,2), color(0.5f, 0.5f, 0.5f)));
}
#endif